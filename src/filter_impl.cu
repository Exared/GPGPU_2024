#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <thread>
#include <cstdio>
#include <cfloat>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        // We don't exit when we encounter CUDA errors in this example.
        std::exit(EXIT_FAILURE);
    }
}

struct rgb {
    uint8_t r, g, b;
};

struct lab {
    float l, a, b;
};

struct xyz {
    float x, y, z;
};

__device__ xyz rgb_to_xyz(rgb rgb) {
    double r = rgb.r / 255.0;
    double g = rgb.g / 255.0;
    double b = rgb.b / 255.0;

    r = r > 0.04045 ? __powf((r + 0.055) / 1.055, 2.4) : r / 12.92;
    g = g > 0.04045 ? __powf((g + 0.055) / 1.055, 2.4) : g / 12.92;
    b = b > 0.04045 ? __powf((b + 0.055) / 1.055, 2.4) : b / 12.92;

    r *= 100.0;
    g *= 100.0;
    b *= 100.0;

    xyz xyz;
    xyz.x = r * 0.4124 + g * 0.3576 + b * 0.1805;
    xyz.y = r * 0.2126 + g * 0.7152 + b * 0.0722;
    xyz.z = r * 0.0193 + g * 0.1192 + b * 0.9505;

    return xyz;
}

__device__ lab xyz_to_lab(xyz xyz) {
    double x = xyz.x / 95.047;
    double y = xyz.y / 100.000;
    double z = xyz.z / 108.883;

    x = x > 0.008856 ? __powf(x, 1.0/3.0) : (7.787 * x) + (16.0 / 116.0);
    y = y > 0.008856 ? __powf(y, 1.0/3.0) : (7.787 * y) + (16.0 / 116.0);
    z = z > 0.008856 ? __powf(z, 1.0/3.0) : (7.787 * z) + (16.0 / 116.0);

    lab lab;
    lab.l = (116.0 * y) - 16.0;
    lab.a = 500.0 * (x - y);
    lab.b = 200.0 * (y - z);

    return lab;
}

__device__ lab rgb_to_lab(rgb rgb) {
    xyz xyz = rgb_to_xyz(rgb);
    return xyz_to_lab(xyz);
}

__global__ void convert_to_cielab(std::byte* buffer, int width, int height, int stridein, int strideout, std::byte* output) {
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= width || y >= height)
        return; 

    rgb* lineptr = (rgb*) (buffer + y * stridein);
    lab* outptr = (lab*) (output + y * strideout);
    outptr[x] = rgb_to_lab(lineptr[x]);
}

__global__ void compute_residual_image(
    std::byte* buffer1, //lab
    std::byte* buffer2, //lab
    std::byte* residual, //float
    int width,
    int height,
    int stride1,//lab stride
    int stride2) { ///float stride
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= width || y >= height)
        return;
    
    lab *lineptr1 = (lab*) (buffer1 + y * stride1);
    lab *lineptr2 = (lab*) (buffer2 + y * stride1);
    float *outptr = (float*) (residual + y * stride2);
    outptr[x] = sqrt(__powf(lineptr1[x].l - lineptr2[x].l, 2) + __powf(lineptr1[x].a - lineptr2[x].a, 2) + __powf(lineptr1[x].b - lineptr2[x].b, 2));
}

// Kernel pour l'érosion
__global__ void erosion_kernel(
    float* input,
    float* output,
    int width,
    int height,
    int stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float minVal = FLT_MAX;

    for (int dy = -3; dy <= 3; ++dy) {
        for (int dx = -3; dx <= 3; ++dx) {
            int ix = x + dx;
            int iy = y + dy;
            if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                float val = input[iy * stride / sizeof(float) + ix];
                minVal = fminf(minVal, val);
            }
        }
    }

    output[y * stride / sizeof(float) + x] = minVal;
}

// Kernel pour la dilatation
__global__ void dilatation_kernel(float* input,
    float* output,
    int width,
    int height,
    int stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float maxVal = -FLT_MAX;

    for (int dy = -3; dy <= 3; ++dy) {
        for (int dx = -3; dx <= 3; ++dx) {
            int ix = x + dx;
            int iy = y + dy;
            if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                float val = input[iy * stride / sizeof(float) + ix];
                maxVal = fmaxf(maxVal, val);
            }
        }
    }

    output[y * stride / sizeof(float) + x] = maxVal;
}

__global__ void hysteresis_threshold_kernel(
    float* dilatation, 
    unsigned char* mask, 
    int width, 
    int height, 
    int dilatation_stride,
    int mask_stride,
    float low_threshold,
    float high_threshold) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float pixelValue = dilatation[y * dilatation_stride / sizeof(float) + x];
    unsigned char maskValue = 0;  // Initialisation à 0 (arrière-plan)

    // Seuillage d'hystérésis
    if (pixelValue > high_threshold) {
        maskValue = 1;  // Premier plan
    } else if (pixelValue > low_threshold) {
        // Vérification des voisins pour les valeurs entre les deux seuils
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                int ix = x + dx;
                int iy = y + dy;
                if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                    float neighborValue = dilatation[iy * dilatation_stride / sizeof(float) + ix];
                    if (neighborValue > high_threshold) {
                        maskValue = 1;  // Premier plan si connecté à un voisin de premier plan
                        break;
                    }
                }
            }
            if (maskValue == 1) break;  // Arrêter la recherche si un voisin de premier plan est trouvé
        }
    }

    mask[y * mask_stride + x] = maskValue;
}

__global__ void apply_mask(
    std::byte* buffer, 
    unsigned char* mask, 
    int width, 
    int height, 
    int buffer_stride,
    int mask_stride) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    rgb* lineptr = (rgb*) (buffer + y * buffer_stride);
    unsigned char* maskptr = (unsigned char*) (mask + y * mask_stride);
    if (maskptr[x] == 1) {
        lineptr[x].r = 255;
    }
}

__global__ void recompute_mean_background(
    std::byte* background, 
    std::byte* buffer,
    int width, 
    int height, 
    int background_stride,
    int buffer_stride,
    int frame_count) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    lab* background_lineptr = (lab*) (background + y * background_stride);
    lab* buffer_lineptr = (lab*) (buffer + y * buffer_stride);
    background_lineptr[x].l = (background_lineptr[x].l * (frame_count - 1) + buffer_lineptr[x].l) / frame_count;
    background_lineptr[x].a = (background_lineptr[x].a * (frame_count - 1) + buffer_lineptr[x].a) / frame_count;
    background_lineptr[x].b = (background_lineptr[x].b * (frame_count - 1) + buffer_lineptr[x].b) / frame_count;
}

int first = 0;
std::byte* first_image_lab;
int frame_count = 0;

extern "C" {
    void filter_impl(uint8_t* src_buffer, int width, int height, int src_stride, int pixel_stride)
    {
        assert(sizeof(rgb) == pixel_stride);
        std::byte* dBuffer;
        size_t pitch;
        std::byte* dlabBuffer;
        size_t labpitch;
        std::byte* dResidual;
        size_t residualpitch;
        std::byte* dErosion;
        size_t erosionpitch;
        std::byte* dDilatation;
        size_t dilatationpitch;
        std::byte* binary_mask;
        size_t binary_mask_pitch;

        hipError_t err;
        frame_count++;

        dim3 blockSize(16,16);
        dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);

        // Allocate a buffer on the GPU for the input
        err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
        CHECK_CUDA_ERROR(err);
        // Copy the input buffer to the GPU buffer
        err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);
        
        if (first == 0) {
            //allocate the first image
            err = hipMallocPitch(&first_image_lab, &labpitch, width * sizeof(lab), height);
            CHECK_CUDA_ERROR(err);
            convert_to_cielab<<<gridSize, blockSize>>>(dBuffer, width, height, pitch, labpitch, first_image_lab);
            first = 1;
            hipFree(dBuffer);
            return;
        }

        // Allocate a buffer on the GPU for the input
        err = hipMallocPitch(&dlabBuffer, &labpitch, width * sizeof(lab), height);
        CHECK_CUDA_ERROR(err);

        convert_to_cielab<<<gridSize, blockSize>>>(dBuffer, width, height, pitch, labpitch, dlabBuffer);

        recompute_mean_background<<<gridSize, blockSize>>>(first_image_lab, dlabBuffer, width, height, labpitch, labpitch, frame_count);

        // Allocate a buffer on the GPU for the residual
        err = hipMallocPitch(&dResidual, &residualpitch, width * sizeof(float), height);
        CHECK_CUDA_ERROR(err);

        compute_residual_image<<<gridSize, blockSize>>>(first_image_lab, dlabBuffer, dResidual, width, height, labpitch, residualpitch);
        
        // Allocate a buffer on the GPU for the erosion
        err = hipMallocPitch(&dErosion, &erosionpitch, width * sizeof(float), height);
        CHECK_CUDA_ERROR(err);

        erosion_kernel<<<gridSize, blockSize>>>((float*)dResidual, (float*)dErosion, width, height, residualpitch);

        // Allocate a buffer on the GPU for the dilatation
        err = hipMallocPitch(&dDilatation, &dilatationpitch, width * sizeof(float), height);
        CHECK_CUDA_ERROR(err);

        dilatation_kernel<<<gridSize, blockSize>>>((float*)dErosion, (float*)dDilatation, width, height, erosionpitch);

        // Allocate a buffer on the GPU for the binary mask
        err = hipMallocPitch(&binary_mask, &binary_mask_pitch, width * sizeof(unsigned char), height);
        CHECK_CUDA_ERROR(err);

        hysteresis_threshold_kernel<<<gridSize, blockSize>>>((float*)dDilatation, (unsigned char*)binary_mask, width, height, dilatationpitch, binary_mask_pitch, 4, 30);

        apply_mask<<<gridSize, blockSize>>>(dBuffer, (unsigned char*)binary_mask, width, height, pitch, binary_mask_pitch);
        
        // Copy the result back to the CPU
        err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch, width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(err);

        hipFree(dBuffer);
        hipFree(dlabBuffer);
        hipFree(dResidual);
        hipFree(dErosion);
        hipFree(dDilatation);
        hipFree(binary_mask);

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);
    }   
}